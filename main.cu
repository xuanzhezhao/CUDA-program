
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    std::cout << "Hello, World!" << std::endl;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for(int i=0;i<deviceCount;i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        std::cout << "======================================================" << std::endl;
        std::cout << "GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "Global GPU RAM: " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "Num of SM:" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小:" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数:" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数:" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数:" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量:" << devProp.multiProcessorCount << std::endl;
        std::cout << "======================================================" << std::endl;}
        return 0;
}
